#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
#define ROWS 32
#define COLS 16
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
__global__ void Kerneltest(int **da, unsigned int rows, unsigned int cols)
{
  unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;
  unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;
  if (row < rows && col < cols)
  {
    da[row][col] = row*cols + col;
  }
}
 
extern "C" int func() // 注意这里定义形式
{
  int **da = NULL;
  int **ha = NULL;
  int *dc = NULL;
  int *hc = NULL;
  hipError_t res;
  int r, c;
  bool is_right=true;
 
  res = hipMalloc((void**)(&da), ROWS*sizeof(int*));CHECK(res)
  res = hipMalloc((void**)(&dc), ROWS*COLS*sizeof(int));CHECK(res)
  ha = (int**)malloc(ROWS*sizeof(int*));
  hc = (int*)malloc(ROWS*COLS*sizeof(int));
 
  for (r = 0; r < ROWS; r++)
  {
    ha[r] = dc + r*COLS;
  }
  res = hipMemcpy((void*)(da), (void*)(ha), ROWS*sizeof(int*), hipMemcpyHostToDevice);CHECK(res)
  dim3 dimBlock(16,16);
  dim3 dimGrid((COLS+dimBlock.x-1)/(dimBlock.x), (ROWS+dimBlock.y-1)/(dimBlock.y));
  Kerneltest<<<dimGrid, dimBlock>>>(da, ROWS, COLS);
  res = hipMemcpy((void*)(hc), (void*)(dc), ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost);CHECK(res)
 
  for (r = 0; r < ROWS; r++)
  {
    for (c = 0; c < COLS; c++)
    {   
      printf("%4d ", hc[r*COLS+c]);
      if (hc[r*COLS+c] != (r*COLS+c))
      {   
        is_right = false;
      }   
    }   
    printf("\n");
  }
  printf("the result is %s!\n", is_right? "right":"false");
 
  hipFree((void*)da);
  hipFree((void*)dc);
  free(ha);
  free(hc);
//  getchar();
  return 0;
}