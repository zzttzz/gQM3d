#include "hip/hip_runtime.h"
#include "CudaSplitBadElement.h"
#include "CudaInsertPoint.h"
#include "CudaMesh.h"
#include <time.h>

// This function splits the encroached segments iteratively
void splitBadElements(
	RealD& t_pointlist,
	TriHandleD& t_point2trilist,
	TetHandleD& t_point2tetlist,
	PointTypeD& t_pointtypelist,
	RealD& t_pointradius,
	IntD& t_seglist,
	TriHandleD& t_seg2trilist,
	TetHandleD& t_seg2tetlist,
	IntD& t_seg2parentidxlist,
	IntD& t_segparentendpointidxlist,
	TriStatusD& t_segstatus,
	IntD& t_trifacelist,
	TetHandleD& t_tri2tetlist,
	TriHandleD& t_tri2trilist,
	TriHandleD& t_tri2seglist,
	IntD& t_tri2parentidxlist,
	IntD& t_triid2parentoffsetlist,
	IntD& t_triparentendpointidxlist,
	TriStatusD& t_tristatus,
	IntD& t_tetlist,
	TetHandleD& t_neighborlist,
	TriHandleD& t_tet2trilist,
	TriHandleD& t_tet2seglist,
	TetStatusD& t_tetstatus,
	IntD& t_segencmarker,
	IntD& t_subfaceencmarker,
	int& numofpoints,
	int& numofsubseg,
	int& numofsubface,
	int& numoftet,
	MESHBH* behavior,
	int debug_msg,
	bool debug_error,
	bool debug_timing
)
{
	IntD t_badeleidlist;
	IntD t_encseglist, t_encsubfacelist, t_badtetlist;
	IntD t_threadmarker;

	TetHandleD t_recordoldtetlist;
	IntD t_recordoldtetidx;

	int numberofbadeles;
	int numberofencsegs, numberofencsubfaces, numberofbadtets;

	clock_t tv[2];
	int npt[2];
	int code = 1;
	int iteration = 0;
	int counter;
	while (true)
	{
		//printf("%d, ", iteration);
		// Update the active bad elements list.
		// Exclude the empty ones (their encroachment markers have already been set to -1).
		numberofencsegs = updateActiveListByMarker_Slot(t_segencmarker, t_encseglist, numofsubseg);
		numberofencsubfaces = updateActiveListByMarker_Slot(t_subfaceencmarker, t_encsubfacelist, numofsubface);
		numberofbadtets = updateActiveListByStatus_Slot(t_tetstatus, t_badtetlist, numoftet);
		if (numberofbadtets == 0)
			break;
		if (numberofbadtets <= behavior->minbadtets && iteration >= behavior->miniter)
		{
			code = 0;
			break;
		}

		numberofbadeles = numberofencsegs + numberofencsubfaces + numberofbadtets;
		if (debug_msg) printf("      Iteration #%d: number of bad elements = %d (#%d segs, #%d subfaces, #%d tets)\n", 
			iteration, numberofbadeles, numberofencsegs, numberofencsubfaces, numberofbadtets);

		t_badeleidlist.resize(numberofbadeles);
		thrust::copy_n(t_encseglist.begin(), numberofencsegs, t_badeleidlist.begin());
		thrust::copy_n(t_encsubfacelist.begin(), numberofencsubfaces, t_badeleidlist.begin() + numberofencsegs);
		thrust::copy_n(t_badtetlist.begin(), numberofbadtets, t_badeleidlist.begin() + numberofencsegs + numberofencsubfaces);

		t_threadmarker.resize(numberofbadeles);
		thrust::fill_n(t_threadmarker.begin(), numberofencsegs, 0);
		thrust::fill_n(t_threadmarker.begin() + numberofencsegs, numberofencsubfaces, 1);
		thrust::fill_n(t_threadmarker.begin() + numberofencsegs + numberofencsubfaces, numberofbadtets, 2);

		//tv[0] = clock();
		//npt[0] = numofpoints;
		// Insert points concurrently
		code =
			insertPoint_New(
				t_pointlist,
				t_point2trilist,
				t_point2tetlist,
				t_pointtypelist,
				t_pointradius,
				t_seglist,
				t_seg2trilist,
				t_seg2tetlist,
				t_seg2parentidxlist,
				t_segparentendpointidxlist,
				t_segstatus,
				t_trifacelist,
				t_tri2tetlist,
				t_tri2trilist,
				t_tri2seglist,
				t_tri2parentidxlist,
				t_triid2parentoffsetlist,
				t_triparentendpointidxlist,
				t_tristatus,
				t_tetlist,
				t_neighborlist,
				t_tet2trilist,
				t_tet2seglist,
				t_tetstatus,
				t_segencmarker,
				t_subfaceencmarker,
				t_badeleidlist,
				t_threadmarker,
				t_recordoldtetlist,
				t_recordoldtetidx,
				numberofbadeles,
				numberofencsegs,
				numberofencsubfaces,
				numberofbadtets,
				numofpoints,
				numofsubseg,
				numofsubface,
				numoftet,
				behavior,
				-1,
				-1,
				iteration,
				debug_msg,
				debug_error,
				debug_timing
			);

		if (!code)
			break;

		hipDeviceSynchronize();

		//if (iteration == 3)
		//	break;

		iteration++;

		//tv[1] = clock();
		//npt[1] = numofpoints;
		//printf("%f, %d\n", (REAL)(tv[1] - tv[0]), npt[1] - npt[0]);
	}
	if (!code && debug_msg)
		printf("      End with %d bad tets\n", numberofbadtets);
}